#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <math.h>
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
  int width;
  int height;
  int stride;
  double* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 32

// Get a matrix element
__device__ double GetElement(const Matrix A, int row, int col)
{
  return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
    double value)
{
  A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
  Matrix Asub;
  Asub.width    = BLOCK_SIZE;
  Asub.height   = BLOCK_SIZE;
  Asub.stride   = A.stride;
  Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
    + BLOCK_SIZE * col];
  return Asub;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
  // Block row and column
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  // Each thread block computes one sub-matrix Csub of C
  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

  // Each thread computes one element of Csub
  // by accumulating results into Cvalue
  double Cvalue = 0;

  // Thread row and column within Csub
  int row = threadIdx.y;
  int col = threadIdx.x;

  // Loop over all the sub-matrices of A and B that are
  // required to compute Csub
  // Multiply each pair of sub-matrices together
  // and accumulate the results
  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

    // Get sub-matrix Asub of A
    Matrix Asub = GetSubMatrix(A, blockRow, m);

    // Get sub-matrix Bsub of B
    Matrix Bsub = GetSubMatrix(B, m, blockCol);

    // Shared memory used to store Asub and Bsub respectively
    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load Asub and Bsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix
    As[row][col] = GetElement(Asub, row, col);
    Bs[row][col] = GetElement(Bsub, row, col);

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();

    // Multiply Asub and Bsub together
    for (int e = 0; e < BLOCK_SIZE; ++e)
      Cvalue += As[row][e] * Bs[e][col];

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write Csub to device memory
  // Each thread writes one element
  SetElement(Csub, row, col, Cvalue);
}
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
  hipEvent_t start, stop, startT, stopT;
  float time,full;
  hipEventCreate(&start);
  hipEventCreate(&startT);
  hipEventCreate(&stop);
  hipEventCreate(&stopT);

  // Load A and B to device memory
  hipEventRecord(startT,0);
  Matrix d_A;
  d_A.width = d_A.stride = A.width; d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(double);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size,
      hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = d_B.stride = B.width; d_B.height = B.height;
  size = B.width * B.height * sizeof(double);

  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size,
      hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = d_C.stride = C.width; d_C.height = C.height;
  size = C.width * C.height * sizeof(double);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

  hipEventRecord(start,0);
  MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  hipEventRecord(stopT,0);
  hipEventSynchronize(stopT);

  hipEventElapsedTime(&time, start, stop);
  hipEventElapsedTime(&full, startT, stopT);
  printf ("Time for the kernel: %f ms\n", time);
  printf ("Time Full: %f ms\n", full);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

void runs(int side) {

  double *ma,*mb,*mc;
  int size = side * side;
  ma = new double[size];
  mb = new double[size];
  mc = new double[size];
  for (int i = 0; i < size; i++) {
    ma[i] = 2; mb[i] = 2; mc[i] = 0;
  }

  Matrix matA,matB,matC;
  matA.width = side;
  matA.height = side;
  matA.elements = ma;
  matB.width = side;
  matB.height = side;
  matB.elements = mb;
  matC.width = side;
  matC.height = side;
  matC.elements = mc;

  MatMul(matA,matB,matC);
}

// run the benchmarks
int main () {
  for (int j = 1; j <= 5; j++) {
    printf("===== Iter = %d ===== \n",j);
    for (int i = 6; i <= 13; i++) {
      int side = pow(2,i);
      printf("Size %d x %d\n", side,side);
      runs(side);
      printf("==========");
    }
  }
  return 0;
}
